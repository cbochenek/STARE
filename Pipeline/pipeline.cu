#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include <vector>
#include <memory>
#include <iostream>
using std::cout;
using std::cerr;
using std::endl;
#include <sstream>
#include <iomanip>
#include <string>
#include <fstream>
#include <time.h>

#include </usr/local/psrdada/mopsr/src/sigproc/sigproc.h>
#include </home/user/sigproc-4.3/header.h>

#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using thrust::host_vector;
using thrust::device_vector;
#include <thrust/version.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/gather.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/fill.h>

#include "hd/pipeline.h"
#include "hd/maths.h"
#include "hd/clean_filterbank_rfi.h"

#include "hd/remove_baseline.h"
#include "hd/matched_filter.h"
#include "hd/get_rms.h"
#include "hd/find_giants.h"
#include "hd/label_candidate_clusters.h"
#include "hd/merge_candidates.h"

#include "hd/DataSource.h"
#include "hd/ClientSocket.h"
#include "hd/SocketException.h"
#include "hd/stopwatch.h"         // For benchmarking
//#include "hd/write_time_series.h" // For debugging

typedef thrust::device_vector<hd_byte>::iterator EIB;
typedef thrust::device_vector<unsigned short>::iterator EIS;
typedef thrust::device_vector<int>::iterator II;

#include <dedisp.h>

FILE *output;

void send_string(char *string) /* includefile */
{
  int len;
  len=strlen(string);
  fwrite(&len, sizeof(int), 1, output);
  fwrite(string, sizeof(char), len, output);
}

void send_float(char *name,float floating_point) /* includefile */
{
  send_string(name);
  fwrite(&floating_point,sizeof(float),1,output);
}

void send_double (char *name, double double_precision) /* includefile */
{
  send_string(name);
  fwrite(&double_precision,sizeof(double),1,output);
}

void send_int(char *name, int integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(int),1,output);
}

void send_char(char *name, char integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(char),1,output);
}


void send_long(char *name, long integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(long),1,output);
}

void send_long_double(char *name, long double long_double) /* includefile */
{
  send_string(name);
  fwrite(&long_double,sizeof(long double),1,output);
}

void send_coords(double raj, double dej, double az, double za) /*includefile*/
{
  if ((raj != 0.0) || (raj != -1.0)) send_double("src_raj",raj);
  if ((dej != 0.0) || (dej != -1.0)) send_double("src_dej",dej);
  if ((az != 0.0)  || (az != -1.0))  send_double("az_start",az);
  if ((za != 0.0)  || (za != -1.0))  send_double("za_start",za);
}

#define HD_BENCHMARK

#ifdef HD_BENCHMARK
  void start_timer(Stopwatch& timer) { timer.start(); }
  void stop_timer(Stopwatch& timer) { hipDeviceSynchronize(); timer.stop(); }
#else
  void start_timer(Stopwatch& timer) { }
  void stop_timer(Stopwatch& timer) { }
#endif // HD_BENCHMARK

#include <utility> // For std::pair
template<typename T, typename U>
std::pair<T&,U&> tie(T& a, U& b) { return std::pair<T&,U&>(a,b); }

struct hd_pipeline_t {
  hd_params   params;
  dedisp_plan dedispersion_plan;
  //MPI_Comm    communicator;

  // Memory buffers used during pipeline execution
  std::vector<hd_byte>    h_clean_filterbank;
  host_vector<hd_byte>    h_dm_series;
  device_vector<hd_float> d_time_series;
  device_vector<hd_float> d_filtered_series;
};

hd_error allocate_gpu(const hd_pipeline pl) {
  // TODO: This is just a simple proc-->GPU heuristic to get us started
  int gpu_count;
  hipGetDeviceCount(&gpu_count);
  //int proc_idx;
  //MPI_Comm comm = pl->communicator;
  //MPI_Comm_rank(comm, &proc_idx);
  int proc_idx = pl->params.beam;
  int gpu_idx = pl->params.gpu_id;
  
  hipError_t cerror = hipSetDevice(gpu_idx);
  if( cerror != hipSuccess ) {
    cerr << "Could not setCudaDevice to " << gpu_idx << ": " << hipGetErrorString(cerror) <<  endl;
    return throw_cuda_error(cerror);
  }
  
  if( pl->params.verbosity >= 1 ) {
    cout << "Process " << proc_idx << " using GPU " << gpu_idx << endl;
  }
  
  if( !pl->params.yield_cpu ) {
    if( pl->params.verbosity >= 2 ) {
      cout << "\tProcess " << proc_idx << " setting CPU to spin" << endl;
    }
    cerror = hipSetDeviceFlags(hipDeviceScheduleSpin);
    if( cerror != hipSuccess ) {
      return throw_cuda_error(cerror);
    }
  }
  else {
    if( pl->params.verbosity >= 2 ) {
      cout << "\tProcess " << proc_idx << " setting CPU to yield" << endl;
    }
    // Note: This Yield flag doesn't seem to work properly.
    //   The BlockingSync flag does the job, although it may interfere
    //     with GPU/CPU overlapping (not currently used).
    //cerror = hipSetDeviceFlags(hipDeviceScheduleYield);
    cerror = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if( cerror != hipSuccess ) {
      return throw_cuda_error(cerror);
    }
  }
  
  return HD_NO_ERROR;
}

unsigned int get_filter_index(unsigned int filter_width) {
  // This function finds log2 of the 32-bit power-of-two number v
  unsigned int v = filter_width;
  static const unsigned int b[] = {0xAAAAAAAA, 0xCCCCCCCC, 0xF0F0F0F0, 
                                   0xFF00FF00, 0xFFFF0000};
  register unsigned int r = (v & b[0]) != 0;
  for( int i=4; i>0; --i) {
    r |= ((v & b[i]) != 0) << i;
  }
  return r;
}

hd_error hd_create_pipeline(hd_pipeline* pipeline_, hd_params params) {
  *pipeline_ = 0;
  
  // Note: We use a smart pointer here to automatically clean up after errors
  typedef std::auto_ptr<hd_pipeline_t> smart_pipeline_ptr;
  smart_pipeline_ptr pipeline = smart_pipeline_ptr(new hd_pipeline_t());
  if( !pipeline.get() ) {
    return throw_error(HD_MEM_ALLOC_FAILED);
  }
  
  pipeline->params = params;
  
  if( params.verbosity >= 2 ) {
    cout << "\tAllocating GPU..." << endl;
  }
  
  hd_error error = allocate_gpu(pipeline.get());
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }
  
  if( params.verbosity >= 1 ) {
    cout << "nchans = " << params.nchans << endl;
    cout << "dt     = " << params.dt << endl;
    cout << "f0     = " << params.f0 << endl;
    cout << "df     = " << params.df << endl;
    //cout << "nsnap     = " << params.nsnap << endl;
  }
  
  if( params.verbosity >= 2 ) {
    cout << "\tCreating dedispersion plan..." << endl;
  }
  
  dedisp_error derror;
  derror = dedisp_create_plan(&pipeline->dedispersion_plan,
                              params.nchans, params.dt,
                              params.f0, params.df);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  // TODO: Consider loading a pre-generated DM list instead for flexibility
  derror = dedisp_generate_dm_list(pipeline->dedispersion_plan,
                                   pipeline->params.dm_min,
                                   pipeline->params.dm_max,
                                   pipeline->params.dm_pulse_width,
                                   pipeline->params.dm_tol);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  
  if( pipeline->params.use_scrunching ) {
    derror = dedisp_enable_adaptive_dt(pipeline->dedispersion_plan,
                                       pipeline->params.dm_pulse_width,
                                       pipeline->params.scrunch_tol);
    if( derror != DEDISP_NO_ERROR ) {
      return throw_dedisp_error(derror);
    }
  }
  
  *pipeline_ = pipeline.release();
  
  if( params.verbosity >= 2 ) {
    cout << "\tInitialisation complete." << endl;
  }
  
  if( params.verbosity >= 1 ) {
    cout << "Using Thrust v"
         << THRUST_MAJOR_VERSION << "."
         << THRUST_MINOR_VERSION << "."
         << THRUST_SUBMINOR_VERSION << endl;
  }
  
  return HD_NO_ERROR;
}

// functor to fill permutation indices
struct permute_functor
{
	
	__device__
	int operator()(const int &x) const {

	    int idx, prod;
	    idx = (int)(x / 8);
	    prod = (int)(x % 8);

	    return (int)(idx*8 + 7-prod);

	}
};

// functor to make indices for summation
struct summing_functor
{

	int n, nsnap;
	summing_functor(int _n, int _nsnap) : n(_n), nsnap(_nsnap) {}
	
	__device__
	int operator()(const int &x) const {

	    int idx = (int)(x / 2048);
	    int chan = (int)(x % 2048);
	    
	    return chan + 2048 * (idx*nsnap + n);

	}
};

hd_error hd_execute(hd_pipeline pl,
                    const hd_byte* h_filterbank, hd_size nsamps, hd_size nbits,
                    hd_size first_idx, hd_size* nsamps_processed) {
  hd_error error = HD_NO_ERROR;

  
  
  Stopwatch total_timer;
  Stopwatch memory_timer;
  Stopwatch clean_timer;
  Stopwatch dedisp_timer;
  Stopwatch communicate_timer;
  Stopwatch copy_timer;
  Stopwatch baseline_timer;
  Stopwatch normalise_timer;
  Stopwatch filter_timer;
  Stopwatch coinc_timer;
  Stopwatch giants_timer;
  Stopwatch candidates_timer;
  Stopwatch write_timer; 
 
  start_timer(total_timer);

  printf("First idx %lu\n",first_idx);

//hipDeviceProp_t  prop;
//hipGetDeviceProperties(&prop,0);
//std::cout << prop.name << std::endl;

  start_timer(clean_timer);
  // Note: Filterbank cleaning must be done out-of-place
  hd_size nbytes = nsamps * pl->params.nchans * nbits / 8;
  start_timer(memory_timer);
  pl->h_clean_filterbank.resize(nbytes);
  std::vector<int>          h_killmask(pl->params.nchans, 1);
  stop_timer(memory_timer);

  // get mjd, and decide whether to look at Crab
  time_t rawtime;
  struct tm *info;
  time(&rawtime);
  info = localtime(&rawtime);
  double daytime = (double)(info->tm_hour+info->tm_min/60.+info->tm_sec/3600.);
  double mjd = (double)(57754.+info->tm_yday+(info->tm_hour+7.)/24.+info->tm_min/(24.*60.)+info->tm_sec/(24.*60.*60.));
  printf("Have MJD %.4lf, DAYTIME %.4lf\n",mjd,daytime);
  //FILE *fcrab;
  double tmjd;
  int doCrab=0;
  /*fcrab=fopen("/usr/local/heimdall/Share/crab_mjds.dat","r");
  while (doCrab==0 && !feof(fcrab)) {
      fscanf(fcrab,"%lf\n",&tmjd);
      if ((mjd-tmjd)*(mjd-tmjd)<(0.5/24.)*(0.5/24.))
      	 doCrab=1;
  }
  if (doCrab)
     cout << "crabbytime!" << endl;
  
  fclose(fcrab);*/  
  // check for midday RFI: 12:30 to 13:30 local
  int shutter=1;      
  /*if (daytime>12.5 && daytime<13.5) shutter=0;
  if (shutter==0) cout << "shuttered" << endl;
  else cout << "un-shuttered" << endl;*/

  /*if( pl->params.verbosity >= 2 ) {
    cout << "\tCleaning 0-DM filterbank..." << endl;
  }
  
  // Start by cleaning up the filterbank based on the zero-DM time series
  hd_float cleaning_dm = 0.f;
  // Note: We only clean the narrowest zero-DM signals; otherwise we
  //         start removing real stuff from higher DMs.
  error = clean_filterbank_rfi(pl->dedispersion_plan,
                               &h_filterbank[0],
                               nsamps,
                               nbits,
                               &pl->h_clean_filterbank[0],
                               &h_killmask[0],
                               cleaning_dm,
                               pl->params.dt,
                               pl->params.baseline_length,
                               pl->params.rfi_tol,
                               pl->params.rfi_min_beams,
                               1);//pl->params.boxcar_max);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }*/

  // do permutation
  // h_filterbank must have size nsamps*nchans*nsnap*nbits

  // from network to host byte order
  /*thrust::device_vector<unsigned char> d_perm_fil(nbytes*pl->params.nsnap);
  unsigned char* h_perm_fil_p;
  h_perm_fil_p = (unsigned char *)malloc(nbytes*pl->params.nsnap);
  thrust::copy(h_filterbank,h_filterbank+nbytes*pl->params.nsnap,d_perm_fil.begin());
  thrust::device_vector<int> d_idx(nbytes*pl->params.nsnap);
  thrust::sequence(d_idx.begin(),d_idx.end());
  thrust::transform(d_idx.begin(),d_idx.end(),d_idx.begin(),permute_functor());
  thrust::permutation_iterator<EIB,II> iter(d_perm_fil.begin(),d_idx.begin());
  thrust::copy(iter,iter+nbytes*pl->params.nsnap,&h_perm_fil_p[0]);

  // sum over SNAPs
  unsigned short* h_sfil = (unsigned short *)h_perm_fil_p;
  thrust::device_vector<unsigned short> d_perm_fil_p(nsamps * pl->params.nchans * pl->params.nsnap);
  thrust::copy(h_sfil,h_sfil+nsamps*pl->params.nchans*pl->params.nsnap,d_perm_fil_p.begin());
  thrust::device_vector<unsigned short> d_fil(nsamps * pl->params.nchans);
  thrust::fill(d_fil.begin(),d_fil.end(),0);
  thrust::device_vector<int> d_idx2(nsamps * pl->params.nchans);
  for (int i=0;i<pl->params.nsnap;i++) {
      thrust::sequence(d_idx2.begin(),d_idx2.end());
      thrust::transform(d_idx2.begin(),d_idx2.end(),d_idx2.begin(),summing_functor(i,pl->params.nsnap));
      thrust::permutation_iterator<EIS,II> iter2(d_perm_fil_p.begin(),d_idx2.begin());
      thrust::transform(iter2,iter2+nsamps*pl->params.nchans,d_fil.begin(),thrust::plus<unsigned short>());
  }
  unsigned short* h_fil;
  h_fil = (unsigned short *)malloc(2*nsamps * pl->params.nchans);
  thrust::copy(d_fil.begin(),d_fil.end(),&h_fil[0]);
  memcpy(&pl->h_clean_filterbank[0],&h_fil[0],nbytes);*/

  std::copy(h_filterbank,h_filterbank+nbytes,pl->h_clean_filterbank.begin());
  
  if( pl->params.verbosity >= 2 ) {
    cout << "Applying manual killmasks" << endl;
  }

  error = apply_manual_killmasks (pl->dedispersion_plan,
                                  &h_killmask[0], 
                                  pl->params.num_channel_zaps,
                                  pl->params.channel_zaps);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }

  hd_size good_chan_count = thrust::reduce(h_killmask.begin(),
                                           h_killmask.end());
  hd_size bad_chan_count = pl->params.nchans - good_chan_count;
  if( pl->params.verbosity >= 2 ) {
    cout << "Bad channel count = " << bad_chan_count << endl;
  }
  
  stop_timer(clean_timer);
  
  if( pl->params.verbosity >= 3 ) {
    /*
    cout << "\tWriting killmask to disk..." << endl;
    std::ofstream killfile("killmask.dat");
    for( size_t i=0; i<h_killmask.size(); ++i ) {
      killfile << h_killmask[i] << "\n";
    }
    killfile.close();
    
    cout << "\tWriting cleaned filterbank to disk..." << endl;
    write_host_filterbank(&pl->h_clean_filterbank[0],
                          pl->params.nchans, nsamps, nbits,
                          pl->params.dt, pl->params.f0, pl->params.df,
                          "clean_filterbank.fil");
    */
  }
  if( pl->params.verbosity >= 2 ) {
    cout << "\tGenerating DM list..." << endl;
  }
  
  if( pl->params.verbosity >= 3 ) {
    cout << "dm_min = " << pl->params.dm_min << endl;
    cout << "dm_max = " << pl->params.dm_max << endl;
    cout << "dm_tol = " << pl->params.dm_tol << endl;
    cout << "dm_pulse_width = " << pl->params.dm_pulse_width << endl;
    cout << "nchans = " << pl->params.nchans << endl;
    cout << "dt = " << pl->params.dt << endl;
    
    cout << "dedisp nchans = " << dedisp_get_channel_count(pl->dedispersion_plan) << endl;
    cout << "dedisp dt = " << dedisp_get_dt(pl->dedispersion_plan) << endl;
    cout << "dedisp f0 = " << dedisp_get_f0(pl->dedispersion_plan) << endl;
    cout << "dedisp df = " << dedisp_get_df(pl->dedispersion_plan) << endl;
  }
  
  hd_size      dm_count = dedisp_get_dm_count(pl->dedispersion_plan);
  const float* dm_list  = dedisp_get_dm_list(pl->dedispersion_plan);
  
  const dedisp_size* scrunch_factors =
    dedisp_get_dt_factors(pl->dedispersion_plan);
  if (pl->params.verbosity >= 3 ) 
  {
    cout << "DM List for " << pl->params.dm_min << " to " << pl->params.dm_max << endl;
    for( hd_size i=0; i<dm_count; ++i ) {
      cout << dm_list[i] << endl;
    }
  }  

  if( pl->params.verbosity >= 2 ) {
    cout << "Scrunch factors:" << endl;
    for( hd_size i=0; i<dm_count; ++i ) {
      cout << scrunch_factors[i] << " ";
    }
    cout << endl;
  }
  
  // Set channel killmask for dedispersion
  dedisp_set_killmask(pl->dedispersion_plan, &h_killmask[0]);
  
  hd_size nsamps_computed  = nsamps - dedisp_get_max_delay(pl->dedispersion_plan);
  hd_size series_stride    = nsamps_computed;
  
  // Report the number of samples that will be properly processed
  *nsamps_processed = nsamps_computed - pl->params.boxcar_max;
  
  if( pl->params.verbosity >= 3 ) {
    cout << "dm_count = " << dm_count << endl;
    cout << "max delay = " << dedisp_get_max_delay(pl->dedispersion_plan) << endl;
    cout << "nsamps_computed = " << nsamps_computed << endl;
  }
  
  hd_size beam = pl->params.beam;
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tAllocating memory for pipeline computations..." << endl;
  }
  
  start_timer(memory_timer);
  
  pl->h_dm_series.resize(series_stride * pl->params.dm_nbits/8 * dm_count);
  pl->d_time_series.resize(series_stride);
  pl->d_filtered_series.resize(series_stride, 0);
  
  stop_timer(memory_timer);
  
  RemoveBaselinePlan          baseline_remover;
  GetRMSPlan                  rms_getter;
  MatchedFilterPlan<hd_float> matched_filter_plan;
  GiantFinder                 giant_finder;
  
  thrust::device_vector<hd_float> d_giant_peaks;
  thrust::device_vector<hd_size>  d_giant_inds;
  thrust::device_vector<hd_size>  d_giant_begins;
  thrust::device_vector<hd_size>  d_giant_ends;
  thrust::device_vector<hd_size>  d_giant_filter_inds;
  thrust::device_vector<hd_size>  d_giant_dm_inds;
  thrust::device_vector<hd_size>  d_giant_members;
  
  typedef thrust::device_ptr<hd_float> dev_float_ptr;
  typedef thrust::device_ptr<hd_size>  dev_size_ptr;
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tDedispersing for DMs " << dm_list[0]
         << " to " << dm_list[dm_count-1] << "..." << endl;
  }
  
  // Dedisperse
  dedisp_error       derror;
  const dedisp_byte* in = &pl->h_clean_filterbank[0];
  dedisp_byte*       out = &pl->h_dm_series[0];
  dedisp_size        in_nbits = nbits;
  dedisp_size        in_stride = pl->params.nchans * in_nbits/8;
  dedisp_size        out_nbits = pl->params.dm_nbits;
  dedisp_size        out_stride = series_stride * out_nbits/8;
  unsigned           flags = 0;
  start_timer(dedisp_timer);
  derror = dedisp_execute_adv(pl->dedispersion_plan, nsamps,
                              in, in_nbits, in_stride,
                              out, out_nbits, out_stride,
                              flags);
  stop_timer(dedisp_timer);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  
  if( beam == 0 && first_idx == 0 ) {
    // TESTING
    //write_host_time_series((unsigned int*)out, nsamps_computed, out_nbits,
    //                       pl->params.dt, "dedispersed_0.tim");
  }
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tBeginning inner pipeline..." << endl;
  }
  
  // TESTING
  hd_size write_dm = 0;
  
  bool too_many_giants = false;
  int notrig = 0;

  // For each DM
  for( hd_size dm_idx=0; dm_idx<dm_count; ++dm_idx ) {
  //if ((dm_list[dm_idx]>53. && dm_list[dm_idx]<60.5) || (dm_list[dm_idx]>100.)) {

    hd_size  cur_dm_scrunch = scrunch_factors[dm_idx];
    hd_size  cur_nsamps  = nsamps_computed / cur_dm_scrunch;
    hd_float cur_dt      = pl->params.dt * cur_dm_scrunch;
    
    // Bail if the candidate rate is too high
    if( too_many_giants ) {
      break;
    }
    
    if( pl->params.verbosity >= 4 ) {
      cout << "dm_idx     = " << dm_idx << endl;
      cout << "scrunch    = " << scrunch_factors[dm_idx] << endl;
      cout << "cur_nsamps = " << cur_nsamps << endl;
      cout << "dt0        = " << pl->params.dt << endl;
      cout << "cur_dt     = " << cur_dt << endl;
        
      cout << "\tBaselining and normalising each beam..." << endl;
    }
    
    hd_float* time_series = thrust::raw_pointer_cast(&pl->d_time_series[0]);
    
    // Copy the time series to the device and convert to floats
    hd_size offset = dm_idx * series_stride * pl->params.dm_nbits/8;
    start_timer(copy_timer);
    switch( pl->params.dm_nbits ) {
    case 8:
      thrust::copy((unsigned char*)&pl->h_dm_series[offset],
                   (unsigned char*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    case 16:
      thrust::copy((unsigned short*)&pl->h_dm_series[offset],
                   (unsigned short*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    case 32:
      // Note: 32-bit implies float, not unsigned int
      thrust::copy((float*)&pl->h_dm_series[offset],
                   (float*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    default:
      return HD_INVALID_NBITS;
    }
    stop_timer(copy_timer);
    
    // Remove the baseline
    // -------------------
    // Note: Divided by 2 to form a smoothing radius
    hd_size nsamps_smooth = hd_size(pl->params.baseline_length /
                                    (2 * cur_dt));
    // Crop the smoothing length in case not enough samples
    start_timer(baseline_timer);
    
    // TESTING
    error = baseline_remover.exec(time_series, cur_nsamps, nsamps_smooth);
    stop_timer(baseline_timer);
    if( error != HD_NO_ERROR ) {
      return throw_error(error);
    }
    
    if( beam == 0 && dm_idx == write_dm && first_idx == 0 ) {
      // TESTING
      //write_device_time_series(time_series, cur_nsamps,
      //                         cur_dt, "baselined.tim");
    }
    // -------------------
    
    // Normalise
    // ---------
    start_timer(normalise_timer);
    hd_float rms = rms_getter.exec(time_series, cur_nsamps);
    thrust::transform(pl->d_time_series.begin(), pl->d_time_series.end(),
                      thrust::make_constant_iterator(hd_float(1.0)/rms),
                      pl->d_time_series.begin(),
                      thrust::multiplies<hd_float>());
    stop_timer(normalise_timer);
    
    if( beam == 0 && dm_idx == write_dm && first_idx == 0 ) {
      // TESTING
      //write_device_time_series(time_series, cur_nsamps,
      //                         cur_dt, "normalised.tim");
    }
    // ---------
    
    // Prepare the boxcar filters
    // --------------------------
    // We can't process the first and last max-filter-width/2 samples
    hd_size rel_boxcar_max = pl->params.boxcar_max/cur_dm_scrunch;
    
    hd_size max_nsamps_filtered = cur_nsamps + 1 - rel_boxcar_max;
    // This is the relative offset into the time series of the filtered data
    hd_size cur_filtered_offset = rel_boxcar_max / 2;
    
    // Create and prepare matched filtering operations
    start_timer(filter_timer);
    // Note: Filter width is relative to the current time resolution
    matched_filter_plan.prep(time_series, cur_nsamps, rel_boxcar_max);
    stop_timer(filter_timer);
    // --------------------------
    
    hd_float* filtered_series = thrust::raw_pointer_cast(&pl->d_filtered_series[0]);
    
    // Note: Filtering is done using a combination of tscrunching and
    //         'proper' boxcar convolution. The parameter min_tscrunch_width
    //         indicates how much of each to do. Raising min_tscrunch_width
    //         increases sensitivity but decreases performance and vice
    //         versa.
    
    // For each boxcar filter
    // Note: We cannot detect pulse widths < current time resolution
    for( hd_size filter_width=cur_dm_scrunch;
         filter_width<=pl->params.boxcar_max/64;
         filter_width*=2 ) {
      hd_size rel_filter_width = filter_width / cur_dm_scrunch;
      hd_size filter_idx = get_filter_index(filter_width);
      
      if( pl->params.verbosity >= 4 ) {
        cout << "Filtering each beam at width of " << filter_width << endl;
      }
      
      // Note: Filter width is relative to the current time resolution
      hd_size rel_min_tscrunch_width = std::max(pl->params.min_tscrunch_width
                                                / cur_dm_scrunch,
                                                hd_size(1));
      hd_size rel_tscrunch_width = std::max(2 * rel_filter_width
                                            / rel_min_tscrunch_width,
                                            hd_size(1));
      // Filter width relative to cur_dm_scrunch AND tscrunch
      hd_size rel_rel_filter_width = rel_filter_width / rel_tscrunch_width;
      
      start_timer(filter_timer);
      
      error = matched_filter_plan.exec(filtered_series,
                                       rel_filter_width,
                                       rel_tscrunch_width);
      
      if( error != HD_NO_ERROR ) {
        return throw_error(error);
      }
      // Divide and round up
      hd_size cur_nsamps_filtered = ((max_nsamps_filtered-1)
                                     / rel_tscrunch_width + 1);
      hd_size cur_scrunch = cur_dm_scrunch * rel_tscrunch_width;
      
      // Normalise the filtered time series (RMS ~ sqrt(time))
      // TODO: Avoid/hide the ugly thrust code?
      //         Consider making it a method of MatchedFilterPlan
      /*
      thrust::constant_iterator<hd_float> 
        norm_val_iter(1.0 / sqrt((hd_float)rel_filter_width));
      thrust::transform(thrust::device_ptr<hd_float>(filtered_series),
                        thrust::device_ptr<hd_float>(filtered_series)
                        + cur_nsamps_filtered,
                        norm_val_iter,
                        thrust::device_ptr<hd_float>(filtered_series),
                        thrust::multiplies<hd_float>());
      */
      // TESTING Proper normalisation
      hd_float rms = rms_getter.exec(filtered_series, cur_nsamps_filtered);
      thrust::transform(thrust::device_ptr<hd_float>(filtered_series),
                        thrust::device_ptr<hd_float>(filtered_series)
                        + cur_nsamps_filtered,
                        thrust::make_constant_iterator(hd_float(1.0)/rms),
                        thrust::device_ptr<hd_float>(filtered_series),
                        thrust::multiplies<hd_float>());

      stop_timer(filter_timer);
      
      if( beam == 0 && dm_idx == write_dm && first_idx == 0 &&
          filter_width == 8 ) {
        // TESTING
        //write_device_time_series(filtered_series,
        //                         cur_nsamps_filtered,
        //                         cur_dt, "filtered.tim");
      }
      
      hd_size prev_giant_count = d_giant_peaks.size();
      
      if( pl->params.verbosity >= 4 ) {
        cout << "Finding giants..." << filter_width << endl;
      }
      
      start_timer(giants_timer);	
      
      error = giant_finder.exec(filtered_series, cur_nsamps_filtered,
                                pl->params.detect_thresh,
                                //pl->params.cand_sep_time,
                                // Note: This was MB's recommendation
                                pl->params.cand_sep_time * rel_rel_filter_width,
                                d_giant_peaks,
                                d_giant_inds,
                                d_giant_begins,
                                d_giant_ends);
      
      if( error != HD_NO_ERROR ) {
        return throw_error(error);
      }

      stop_timer(giants_timer);					

      hd_size rel_cur_filtered_offset = (cur_filtered_offset /
                                         rel_tscrunch_width);
      
      using namespace thrust::placeholders;
      thrust::transform(d_giant_inds.begin()+prev_giant_count,
                        d_giant_inds.end(),
                        d_giant_inds.begin()+prev_giant_count,
                        /*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      thrust::transform(d_giant_begins.begin()+prev_giant_count,
                        d_giant_begins.end(),
                        d_giant_begins.begin()+prev_giant_count,
                        /*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      thrust::transform(d_giant_ends.begin()+prev_giant_count,
                        d_giant_ends.end(),
                        d_giant_ends.begin()+prev_giant_count,
                        /*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      
      d_giant_filter_inds.resize(d_giant_peaks.size(), filter_idx);
      d_giant_dm_inds.resize(d_giant_peaks.size(), dm_idx);
      // Note: This could be used to track total member samples if desired
      d_giant_members.resize(d_giant_peaks.size(), 1);

      // Bail if the candidate rate is too high
      hd_size total_giant_count = d_giant_peaks.size();
      hd_float data_length_mins = nsamps * pl->params.dt / 60.0;
      if ( pl->params.max_giant_rate && ( total_giant_count / data_length_mins > pl->params.max_giant_rate ) ) {
        too_many_giants = true;
        float searched = ((float) dm_idx * 100) / (float) dm_count;
	notrig = 1;
        cout << "WARNING: exceeded max giants/min, DM [" << dm_list[dm_idx] << "] space searched " << searched << "%" << endl;
        break;
      }
      start_timer(write_timer);
      if (total_timer.getTime() > 6.5) {
        too_many_giants = true;
	float searched = ((float) dm_idx * 100) / (float) dm_count;
	cout << "WARNING: exceeded max giants processed in 6.5s, DM [" << dm_list[dm_idx] << "] space searched " << searched << "%" << endl;
	break;
      }
      
    } // End of filter width loop
  //} // end of if statement selecting DMs  
  } // End of DM loop

  hd_size giant_count = d_giant_peaks.size();
//  if( pl->params.verbosity >= 2 ) {
    cout << "Giant count = " << giant_count << endl;
//  }
  
  start_timer(candidates_timer);

  thrust::host_vector<hd_float> h_group_peaks;
  thrust::host_vector<hd_size>  h_group_inds;
  thrust::host_vector<hd_size>  h_group_begins;
  thrust::host_vector<hd_size>  h_group_ends;
  thrust::host_vector<hd_size>  h_group_filter_inds;
  thrust::host_vector<hd_size>  h_group_dm_inds;
  thrust::host_vector<hd_size>  h_group_members;
  thrust::host_vector<hd_float> h_group_dms;

  //if (!too_many_giants)
  //{
    thrust::device_vector<hd_size> d_giant_labels(giant_count);
    hd_size* d_giant_labels_ptr = thrust::raw_pointer_cast(&d_giant_labels[0]);
  
    RawCandidates d_giants;
    d_giants.peaks = thrust::raw_pointer_cast(&d_giant_peaks[0]);
    d_giants.inds = thrust::raw_pointer_cast(&d_giant_inds[0]);
    d_giants.begins = thrust::raw_pointer_cast(&d_giant_begins[0]);
    d_giants.ends = thrust::raw_pointer_cast(&d_giant_ends[0]);
    d_giants.filter_inds = thrust::raw_pointer_cast(&d_giant_filter_inds[0]);
    d_giants.dm_inds = thrust::raw_pointer_cast(&d_giant_dm_inds[0]);
    d_giants.members = thrust::raw_pointer_cast(&d_giant_members[0]);
  
    hd_size filter_count = get_filter_index(pl->params.boxcar_max) + 1;

    if( pl->params.verbosity >= 2 ) {
      cout << "Grouping coincident candidates..." << endl;
    }
  
    hd_size label_count;
    error = label_candidate_clusters(giant_count,
                                     *(ConstRawCandidates*)&d_giants,
                                     pl->params.cand_sep_time,
                                     pl->params.cand_sep_filter,
                                     pl->params.cand_sep_dm,
                                     d_giant_labels_ptr,
                                     &label_count);
    if( error != HD_NO_ERROR ) {
      return throw_error(error);
    }
  
    hd_size group_count = label_count;
    if( pl->params.verbosity >= 2 ) {
      cout << "Candidate count = " << group_count << endl;
    }
  
    thrust::device_vector<hd_float> d_group_peaks(group_count);
    thrust::device_vector<hd_size>  d_group_inds(group_count);
    thrust::device_vector<hd_size>  d_group_begins(group_count);
    thrust::device_vector<hd_size>  d_group_ends(group_count);
    thrust::device_vector<hd_size>  d_group_filter_inds(group_count);
    thrust::device_vector<hd_size>  d_group_dm_inds(group_count);
    thrust::device_vector<hd_size>  d_group_members(group_count);
  
    thrust::device_vector<hd_float> d_group_dms(group_count);
  
    RawCandidates d_groups;
    d_groups.peaks = thrust::raw_pointer_cast(&d_group_peaks[0]);
    d_groups.inds = thrust::raw_pointer_cast(&d_group_inds[0]);
    d_groups.begins = thrust::raw_pointer_cast(&d_group_begins[0]);
    d_groups.ends = thrust::raw_pointer_cast(&d_group_ends[0]);
    d_groups.filter_inds = thrust::raw_pointer_cast(&d_group_filter_inds[0]);
    d_groups.dm_inds = thrust::raw_pointer_cast(&d_group_dm_inds[0]);
    d_groups.members = thrust::raw_pointer_cast(&d_group_members[0]);
  
    merge_candidates(giant_count,
                     d_giant_labels_ptr,
                     *(ConstRawCandidates*)&d_giants,
                     d_groups);
  
    // Look up the actual DM of each group
    thrust::device_vector<hd_float> d_dm_list(dm_list, dm_list+dm_count);
    thrust::gather(d_group_dm_inds.begin(), d_group_dm_inds.end(),
                   d_dm_list.begin(),
                   d_group_dms.begin());
  
    // Device to host transfer of candidates
    h_group_peaks = d_group_peaks;
    h_group_inds = d_group_inds;
    h_group_begins = d_group_begins;
    h_group_ends = d_group_ends;
    h_group_filter_inds = d_group_filter_inds;
    h_group_dm_inds = d_group_dm_inds;
    h_group_members = d_group_members;
    h_group_dms = d_group_dms;
    //h_group_flags = d_group_flags;
  //}
  
  if( pl->params.verbosity >= 2 ) {
    cout << "Writing output candidates, utc_start=" << pl->params.utc_start << endl;
  }
  // input data block HDU key
  key_t in_key = 0x0000eada;
  dada_hdu_t* hdu_in = 0;
  multilog_t* log = 0;
  uint64_t header_size = 0;
  hdu_in  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    fprintf (stderr, "dsaX_spectrometer_reorder: could not connect to dada buffer\n");
    return EXIT_FAILURE;
  }
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      //dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }
  long double MJD;
  int MJD_date;
  long double MJD_time;
  double MJD_time_d;
   if (ascii_header_get (header_in, "MJD_START", "%Lf", &MJD) != 1)
    {
      long double MJD = (long double)(57754.+info->tm_yday+(info->tm_hour+8.)/24.+info->tm_min/(24.*60.)+info->tm_sec/(24.*60.*60.));
      //MJD = (char*)(&MJD_double);
      //multilog(log, LOG_WARNING, "Header with no MJD_START. Setting to %s\n", &MJD);
    }
MJD_date = (int)MJD;
MJD_time = MJD - (long double)MJD_date;
int MJD_hour = (long double)MJD_time*24;
int MJD_minute = ((long double)MJD_time*24-MJD_hour)*60;
double MJD_second = (((long double)MJD_time*24-MJD_hour)*60 - MJD_minute)*60;
MJD_time_d = (double)MJD_time;
//printf("MJD: %0.20Lf\n MJD_date: %i\n MJD_time: %0.20Lf\n MJD_time_d: %0.20f MJD_hour: %i\n MJD_minute: %i\n MJD_second: %0.20f",MJD,MJD_date,MJD_time,MJD_time_d,MJD_hour,MJD_minute,MJD_second);
  char buffer[64];
  time_t now = pl->params.utc_start + (time_t) (first_idx / pl->params.spectra_per_second);
  strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&now));

  std::stringstream ss;
  ss << std::setw(2) << std::setfill('0') << (pl->params.beam)%13+1;

  std::ostringstream oss;

  if ( pl->params.coincidencer_host != NULL && pl->params.coincidencer_port != -1 )
  {
    try 
    {
      ClientSocket client_socket ( pl->params.coincidencer_host, pl->params.coincidencer_port );

      strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&(pl->params.utc_start)));

      oss <<  buffer << " ";

      time_t now = pl->params.utc_start + (time_t) (first_idx / pl->params.spectra_per_second);
      strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&now));
      oss << buffer << " ";

      oss << first_idx << " ";
      oss << ss.str() << " ";
      oss << h_group_peaks.size() << endl;
      client_socket << oss.str();
      oss.flush();
      oss.str("");

      for (hd_size i=0; i<h_group_peaks.size(); ++i ) 
      {
        hd_size samp_idx = first_idx + h_group_inds[i];
        oss << h_group_peaks[i] << "\t"
                      << samp_idx << "\t"
                      << samp_idx * pl->params.dt << "\t"
                      << h_group_filter_inds[i] << "\t"
                      << h_group_dm_inds[i] << "\t"
                      << h_group_dms[i] << "\t"
                      << h_group_members[i] << "\t"
                      << first_idx + h_group_begins[i] << "\t"
                      << first_idx + h_group_ends[i] << endl;

        client_socket << oss.str();
        oss.flush();
        oss.str("");
      }
      // client_socket should close when it goes out of scope...
    }
    catch (SocketException& e )
    {
      std::cerr << "SocketException was caught:" << e.description() << "\n";
    }

  }
  //else
  //{

    // HACK %13

    if( pl->params.verbosity >= 2 )
      cout << "Output timestamp: " << buffer << endl;

    //std::string filename = std::string(pl->params.output_dir) + "/" + std::string(buffer) + "_" + ss.str() + ".cand";

   // if( pl->params.verbosity >= 2 )
   //   cout << "Output filename: " << filename << endl;

   FILE *cands_out;
   char ofile[200];
   sprintf(ofile,"%s/heimdall_2.cand",pl->params.output_dir);
   cands_out = fopen(ofile,"a");
   FILE *hit_rate;
   char ofile2[200];
   sprintf(ofile2,"%s/hitrate_2.txt",pl->params.output_dir);
   hit_rate = fopen(ofile2,"a");
    //std::ofstream cand_file(filename.c_str(), std::ios::out);
    //if( pl->params.verbosity >= 2 )
    //  cout << "Dumping " << h_group_peaks.size() << " candidates to " << filename << endl;

    // FILE WRITING VR
    float dm, snr;
    char cmd[300];
    hd_size rawsample;
    int samp, wid;
    char filname[200];
    int s1, s2;

    int maxI=-1;
    float maxSNR=0.;

    std::vector<hd_byte> output_data;
    int sent=0;
    hd_size samp_idx;
 float thresh =7.3;
int s1s [h_group_peaks.size()];
int s2s [h_group_peaks.size()];
for(int i = 0; i<h_group_peaks.size(); i++) {
	s1s[i] = 0;
	s2s[i] = 0;
}
int hit_count = 0;
      for( hd_size i=0; i<h_group_peaks.size(); ++i ) {
        samp_idx = first_idx + h_group_inds[i];
        if ((shutter) && (samp_idx>100000) && ((h_group_peaks[i] >= thresh && h_group_dms[i] > 0. && notrig==0) || (h_group_peaks[i]>7. && h_group_dms[i]>55.0 && h_group_dms[i]<58.5 && notrig==0 && doCrab))) {
	long double cand_time = MJD+samp_idx * pl->params.dt/3600./24.;
	int cand_MJD = (int)cand_time;
	long double cand_hr_dbl = (cand_time-cand_MJD)*24;
	int cand_hour = (cand_time-cand_MJD)*24;
	long double cand_min_dbl = (cand_hr_dbl-cand_hour)*60.;
	int cand_minute = (int) cand_min_dbl;
	long double cand_sec = (cand_min_dbl - cand_minute)*60;
	hit_count++;
	fprintf(cands_out,"%g %lu %g %d %d %g %d %0.20Lf %i %i %i %.12Lf \n",h_group_peaks[i],samp_idx,samp_idx * pl->params.dt,h_group_filter_inds[i],h_group_dm_inds[i],h_group_dms[i],h_group_members[i],MJD,cand_MJD,cand_hour,cand_minute,cand_sec);

	   maxSNR = h_group_peaks[i];
	   maxI = i;
	   //thresh = h_group_peaks[i];
	   samp_idx = first_idx + h_group_inds[i];
	}
        /*cand_file << h_group_peaks[i] << "\t"
                  << samp_idx << "\t"
                  << samp_idx * pl->params.dt << "\t"
                  << h_group_filter_inds[i] << "\t"
                  << h_group_dm_inds[i] << "\t"
                  << h_group_dms[i] << "\t"
                  //<< h_group_flags[i] << "\t"
                  << h_group_members[i] << "\t"
                  // HACK %13
                  //<< (beam+pl->params.beam)%13+1 << "\t"
                  << first_idx + h_group_begins[i] << "\t"
                  << first_idx + h_group_ends[i] << "\t"
                  << "\n";*/

      //}

      if (h_group_peaks.size()>0 && maxI!=-1) {
      samp_idx = first_idx + h_group_begins[maxI];
      if ((shutter) && (samp_idx>100000) && ((h_group_peaks[maxI] >= thresh && h_group_dms[maxI] > 0. && notrig==0) || (h_group_peaks[maxI]>7. && h_group_dms[maxI]>55.0 && h_group_dms[maxI]<58.5 && notrig==0 && doCrab))) {

	   rawsample = (samp_idx-763); // VR wuz hre - edit for different stuff
	   /*sprintf(cmd,"echo %lu | nc -4u -w1 10.10.1.7 11223 &",rawsample);
	   cout << "Sending to dsa1: " << cmd << endl;
	   system(cmd);
	   sprintf(cmd,"echo %lu | nc -4u -w1 10.10.1.8 11223 &",rawsample);
	   system(cmd);
	   sprintf(cmd,"echo %lu | nc -4u -w1 10.10.1.9 11223 &",rawsample);
	   system(cmd);
	   sprintf(cmd,"echo %lu | nc -4u -w1 10.10.1.10 11223 &",rawsample);
	   system(cmd);
	   sprintf(cmd,"echo %lu | nc -4u -w1 10.10.1.11 11223 &",rawsample);
	   system(cmd);*/
//	   sent=1;
	   int start_idx;
	   int end_idx;
	   s1= h_group_inds[maxI]-1000;
           if (s1<0) s1=0;
           s2 = h_group_inds[maxI]+int((0.000761*h_group_dms[maxI])/pl->params.dt)+1000+(int)(pow(2.,h_group_filter_inds[maxI]));
           if (s2>nbytes/(pl->params.nchans*nbits/8)) s2=nbytes/(pl->params.nchans*nbits/8);
	   int duplicate = 0;
	   for(int j=0; j < h_group_peaks.size(); j++)  {
	   	if(s1 == s1s[j]) {
			if(s2 == s2s[j]) {
				duplicate = 1;
			}
           	}
	   }
	if(duplicate == 0) {
	   s1s[i] = s1;
	   s2s[i] = s2;
	   start_idx = s1+first_idx;
	   end_idx = s2+first_idx;
     	   sprintf(filname,"/home/user/candidates/candidate_%d.fil",first_idx+h_group_inds[maxI]);
     	   output = fopen(filname,"wb");
     	   send_string("HEADER_START");
    	   //send_string("source_name");
     	   send_int("machine_id",1);
     	   send_int("telescope_id",82);
     	   send_int("data_type",1); // filterbank data
	   //send_long_double("start_MJD",MJD);
	   send_int("start_sample", start_idx);
	   send_int("end_sample", end_idx);
	   send_int("cand_location", first_idx+h_group_inds[maxI]);
     	   send_double("fch1",pl->params.f0);
     	   send_double("foff",pl->params.df);
     	   send_int("nchans",pl->params.nchans);
     	   send_int("nbits",nbits);
     	   send_double("tstart",MJD_time_d);
	   send_int("MJD_hour",MJD_hour);
           send_int("MJD_minute",MJD_minute);
           send_double("MJD_second",MJD_second);
	   send_int("MJD_start", MJD_date);
     	   send_double("tsamp",pl->params.dt);
     	   send_int("nifs",1);
     	   send_string("HEADER_END");

	   if (s1<0) s1=0;
	   s2 = h_group_inds[maxI]+int((0.000761*h_group_dms[maxI])/pl->params.dt)+1000+(int)(pow(2.,h_group_filter_inds[maxI]));
	   if (s2>nbytes/(pl->params.nchans*nbits/8)) s2=nbytes/(pl->params.nchans*nbits/8);

	   cout << "Outputting data in samples " << s1 << " " << s2 << endl;

	   output_data.resize((s2-s1)*(pl->params.nchans*nbits/8));
	   std::copy(pl->h_clean_filterbank.begin()+s1*(pl->params.nchans*nbits/8),pl->h_clean_filterbank.begin()+s2*(pl->params.nchans*nbits/8),output_data.begin());

     	   fwrite((&output_data[0]),nbits/8,pl->params.nchans*(s2-s1),output);
     	   fclose(output);
	}
      }
}
}

long double MJD_hit = MJD + first_idx * pl->params.dt/3600./24.;
int MJD_date_hit = (int)MJD_hit;
long double MJD_time_hit = MJD_hit - (long double)MJD_date_hit;
int MJD_hour_hit = (long double)MJD_time_hit*24;
int MJD_minute_hit = ((long double)MJD_time_hit*24-MJD_hour_hit)*60;
double MJD_second_hit = (((long double)MJD_time_hit*24-MJD_hour_hit)*60 - MJD_minute_hit)*60;
fprintf(hit_rate,"%i %i %i %f %i\n",MJD_date_hit, MJD_hour_hit, MJD_minute_hit, MJD_second_hit, hit_count);
fclose(hit_rate);
  fclose(cands_out);
  stop_timer(candidates_timer);

/*  cout << "sending first_idx to dsa1" << endl;
  sprintf(cmd,"echo p %lu | nc -4u -w1 10.10.1.7 11223",first_idx);
  system(cmd);
  cout << "sending first_idx to dsa2" << endl;
  sprintf(cmd,"echo p %lu | nc -4u -w1 10.10.1.8 11223",first_idx);
  system(cmd);
  cout << "sending first_idx to dsa3" << endl;
  sprintf(cmd,"echo p %lu | nc -4u -w1 10.10.1.9 11223",first_idx);	
  system(cmd);
  cout << "sending first_idx to dsa4" << endl;
  sprintf(cmd,"echo p %lu | nc -4u -w1 10.10.1.10 11223",first_idx);
  system(cmd);
  cout << "sending first_idx to dsa5" << endl;
  sprintf(cmd,"echo p %lu | nc -4u -w1 10.10.1.11 11223",first_idx);
  system(cmd);*/

  stop_timer(total_timer);
  stop_timer(write_timer);
  
#ifdef HD_BENCHMARK
  if( pl->params.verbosity >= 1 )
  {
  cout << "Mem alloc time:          " << memory_timer.getTime() << endl;
  cout << "0-DM cleaning time:      " << clean_timer.getTime() << endl;
  cout << "Dedispersion time:       " << dedisp_timer.getTime() << endl;
  cout << "Copy time:               " << copy_timer.getTime() << endl;
  cout << "Baselining time:         " << baseline_timer.getTime() << endl;
  cout << "Normalisation time:      " << normalise_timer.getTime() << endl;
  cout << "Filtering time:          " << filter_timer.getTime() << endl;
  cout << "Find giants time:        " << giants_timer.getTime() << endl;
  cout << "Process candidates time: " << candidates_timer.getTime() << endl;
  cout << "Write to disk time:      " << write_timer.getTime() << endl;
  cout << "Total time:              " << total_timer.getTime() << endl;
  }

  hd_float time_sum = (memory_timer.getTime() +
                       clean_timer.getTime() +
                       dedisp_timer.getTime() +
                       copy_timer.getTime() +
                       baseline_timer.getTime() +
                       normalise_timer.getTime() +
                       filter_timer.getTime() +
                       giants_timer.getTime() +
                       candidates_timer.getTime());
  hd_float misc_time = total_timer.getTime() - time_sum;
  
  /*
  std::ofstream timing_file("timing.dat", std::ios::app);
  timing_file << total_timer.getTime() << "\t"
              << misc_time << "\t"
              << memory_timer.getTime() << "\t"
              << clean_timer.getTime() << "\t"
              << dedisp_timer.getTime() << "\t"
              << copy_timer.getTime() << "\t"
              << baseline_timer.getTime() << "\t"
              << normalise_timer.getTime() << "\t"
              << filter_timer.getTime() << "\t"
              << giants_timer.getTime() << "\t"
              << candidates_timer.getTime() << endl;
  timing_file.close();
  */



#endif // HD_BENCHMARK
  
  if( too_many_giants ) {
    return HD_TOO_MANY_EVENTS;
  }
  else {
    return HD_NO_ERROR;
  }

  

  //free(h_perm_fil_p);
  //free(h_fil);

}

void hd_destroy_pipeline(hd_pipeline pipeline) {
  if( pipeline->params.verbosity >= 2 ) {
    cout << "\tDeleting pipeline object..." << endl;
  }
  
  dedisp_destroy_plan(pipeline->dedispersion_plan);
  
  // Note: This assumes memory owned by pipeline cleans itself up
  if( pipeline ) {
    delete pipeline;
  }
}
